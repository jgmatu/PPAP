#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
/*
#define N 512
__global__ void dotProd( int *a, int *b, int *c ) {
      __shared__ int temp[N];

      temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

      __syncthreads(); // Evita condición de carrera.
      if( 0 == threadIdx.x ) {
            int sum = 0;
            for(int i = 0; i < N; i++ ) {
                  sum += temp[i]; //lento
            }
            *c = sum;
      }
}

#define N 2048
#define THREADS_PER_BLOCK 512

__global__ void dotProd( int *a, int *b, int *c ) {
      __shared__ int temp[THREADS_PER_BLOCK];
      int index = threadIdx.x + blockIdx.x * blockDim.x;

      temp[threadIdx.x] = a[index] * b[index];
      __syncthreads(); // Hasta que no rellenen todos los thread temp no puedo continuar...

      if(threadIdx.x == 0) {
            int sum = 0;
            for( int i= 0; i < THREADS_PER_BLOCK; i++ ) {
                  sum += temp[i];
            }
            c[blockIdx.x] = sum;
      }
}
*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;

__global__ void mult(int *a, int *b, int *c)
{
      int pos = threadIdx.x + blockDim.x * blockIdx.x;
      if (pos >= N) return;

      c[pos] = a[pos] * b[pos];
}

__global__ void shared_mult(int *a, int *b, int *c)
{
      __shared__ int mem[THREADS_PER_BLOCK];
      int pos = threadIdx.x + blockIdx.x * blockDim.x;
      mem[threadIdx.x] = a[pos]  * b[pos];

      __syncthreads();
      c[pos] = mem[threadIdx.x];
}

int main(int argc, char const *argv[]) {
      int *a, *b, *c;
      int *dev_a, *dev_b, *dev_c;
      int size = sizeof(int) * N;

      a = (int *) malloc(size);
      b = (int *) malloc(size);
      c = (int *) malloc(size);

      for (int i = 0; i < N ; i++) {
            a[i] = b[i] = 3;
      }

      hipMalloc(&dev_a, size);
      hipMalloc(&dev_b, size);
      hipMalloc(&dev_c, size);

      hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
      hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
      hipMemset(dev_c, 0, size);

      shared_mult<<<(N - 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);

      hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

      for (int i = 0; i < N ; i++) {
            fprintf(stdout, "Numb : %d\n", c[i]);
      }
      hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
      return 0;
}
